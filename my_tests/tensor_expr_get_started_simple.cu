#include "hip/hip_runtime.h"
#include "tvm/runtime/c_runtime_api.h"
#include "tvm/runtime/c_backend_api.h"
void* __tvm_module_ctx = NULL;
static void* __tvm_set_device_packed = NULL;
static void* myadd_kernel0_packed = NULL;
#ifdef __cplusplus
extern "C"
#endif
TVM_DLL int32_t myadd(void* args, void* arg_type_ids, int32_t num_args, void* out_ret_value, void* out_ret_tcode, void* resource_handle) {
  TVMValue stack[5];
  void* stack_tcode = stack;
  TVMValue stack1[10];
  void* stack_value = stack1;
  void* arg0 = (((TVMValue*)args)[0].v_handle);
  int32_t arg0_code = ((int32_t*)arg_type_ids)[(0)];
  void* arg1 = (((TVMValue*)args)[1].v_handle);
  int32_t arg1_code = ((int32_t*)arg_type_ids)[(1)];
  void* arg2 = (((TVMValue*)args)[2].v_handle);
  int32_t arg2_code = ((int32_t*)arg_type_ids)[(2)];
  void* A = (((DLTensor*)arg0)[0].data);
  void* arg0_shape = (((DLTensor*)arg0)[0].shape);
  int32_t n = ((int32_t)((int64_t*)arg0_shape)[(0)]);
  void* arg0_strides = (((DLTensor*)arg0)[0].strides);
  int32_t stride = ((n == 1) ? 0 : ((arg0_strides == NULL) ? 1 : ((int32_t)((int64_t*)arg0_strides)[(0)])));
  int32_t dev_id = (((DLTensor*)arg0)[0].ctx.device_id);
  void* B = (((DLTensor*)arg1)[0].data);
  void* arg1_shape = (((DLTensor*)arg1)[0].shape);
  void* arg1_strides = (((DLTensor*)arg1)[0].strides);
  int32_t stride1 = ((n == 1) ? 0 : ((arg1_strides == NULL) ? 1 : ((int32_t)((int64_t*)arg1_strides)[(0)])));
  void* C = (((DLTensor*)arg2)[0].data);
  void* arg2_shape = (((DLTensor*)arg2)[0].shape);
  void* arg2_strides = (((DLTensor*)arg2)[0].strides);
  int32_t stride2 = ((n == 1) ? 0 : ((arg2_strides == NULL) ? 1 : ((int32_t)((int64_t*)arg2_strides)[(0)])));
  (((TVMValue*)stack_value)[0].v_int64) = ((int64_t)2);
  ((int32_t*)stack_tcode)[(0)] = 0;
  (((TVMValue*)stack_value)[1].v_int64) = ((int64_t)dev_id);
  ((int32_t*)stack_tcode)[(1)] = 0;
  if (__tvm_set_device_packed == NULL) {
    if (TVMBackendGetFuncFromEnv(__tvm_module_ctx, "__tvm_set_device", &__tvm_set_device_packed) != 0) {
      return -1;
    }
  }
  TVMValue ret_val;
  int ret_type_code;
  if (TVMFuncCall(__tvm_set_device_packed, (TVMValue*) stack_value, (int*) stack_tcode, 2, &ret_val, &ret_type_code) != 0) {
    return -1;
  }
  (((TVMValue*)stack_value)[0].v_handle) = C;
  ((int32_t*)stack_tcode)[(0)] = 3;
  (((TVMValue*)stack_value)[1].v_handle) = A;
  ((int32_t*)stack_tcode)[(1)] = 3;
  (((TVMValue*)stack_value)[2].v_handle) = B;
  ((int32_t*)stack_tcode)[(2)] = 3;
  (((TVMValue*)stack_value)[3].v_int64) = ((int64_t)n);
  ((int32_t*)stack_tcode)[(3)] = 0;
  (((TVMValue*)stack_value)[4].v_int64) = ((int64_t)stride);
  ((int32_t*)stack_tcode)[(4)] = 0;
  (((TVMValue*)stack_value)[5].v_int64) = ((int64_t)stride1);
  ((int32_t*)stack_tcode)[(5)] = 0;
  (((TVMValue*)stack_value)[6].v_int64) = ((int64_t)stride2);
  ((int32_t*)stack_tcode)[(6)] = 0;
  (((TVMValue*)stack_value)[7].v_int64) = ((int64_t)((n + 63) >> 6));
  ((int32_t*)stack_tcode)[(7)] = 0;
  (((TVMValue*)stack_value)[8].v_int64) = ((int64_t)64);
  ((int32_t*)stack_tcode)[(8)] = 0;
  if (myadd_kernel0_packed == NULL) {
    if (TVMBackendGetFuncFromEnv(__tvm_module_ctx, "myadd_kernel0", &myadd_kernel0_packed) != 0) {
      return -1;
    }
  }
  TVMValue ret_val1;
  int ret_type_code1;
  if (TVMFuncCall(myadd_kernel0_packed, (TVMValue*) stack_value, (int*) stack_tcode, 9, &ret_val1, &ret_type_code1) != 0) {
    return -1;
  }
  return 0;
}


extern "C" __global__ void myadd_kernel0(float* __restrict__ C, float* __restrict__ A, float* __restrict__ B, int n, int stride, int stride1, int stride2) {
  if (((int)blockIdx.x) < (n >> 6)) {
    C[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) * stride2))] = (A[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) * stride))] + B[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) * stride1))]);
  } else {
    if (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) < n) {
      C[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) * stride2))] = (A[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) * stride))] + B[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) * stride1))]);
    }
  }
}

int main() {
}

