
#include <hip/hip_runtime.h>
extern "C" __global__ void fused_nn_dense_add_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, volatile unsigned* __cuda_kelvin_flag) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 4; ++k_outer) {
    T_dense_rf[(0)] = (T_dense_rf[(0)] + (placeholder[(((k_outer * 64) + ((int)threadIdx.x)))] * placeholder1[((((((int)blockIdx.x) * 256) + (k_outer * 64)) + ((int)threadIdx.x)))]));
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = ((volatile float*)red_buf0)[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
__cuda_kelvin_exit: if (threadIdx.x == 0 & __cuda_kelvin_flag != nullptr) atomicAdd((unsigned*)__cuda_kelvin_flag, 1);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, volatile unsigned* __cuda_kelvin_flag) {
  float compute[8];
  __shared__ float pad_temp_shared[224];
  __shared__ float placeholder_shared[128];
  #pragma unroll
  for (int ff_init = 0; ff_init < 4; ++ff_init) {
    #pragma unroll
    for (int yy_init = 0; yy_init < 2; ++yy_init) {
      compute[(((ff_init * 2) + yy_init))] = 0.000000e+00f;
    }
  }
  for (int ry_outer = 0; ry_outer < 5; ++ry_outer) {
    #pragma unroll
    for (int rx_outer = 0; rx_outer < 5; ++rx_outer) {
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        pad_temp_shared[((((((int)threadIdx.z) * 56) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((2 <= (((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 28) / 14)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 28) / 14)) + ry_outer) < 16)) && (2 <= (rx_outer + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 14)))) && ((rx_outer + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 14)) < 16)) ? placeholder[((((((((((int)threadIdx.z) * 392) + ((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 28) * 196)) + (((int)blockIdx.y) * 28)) + (ry_outer * 14)) + rx_outer) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 28)) - 30))] : 0.000000e+00f);
      }
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
        if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 3)) < 16) {
          if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 128) {
            if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 32) {
              placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)threadIdx.z) * 800) + (((int)threadIdx.x) * 75)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 * 25)) + (ry_outer * 5)) + rx_outer))];
            }
          }
        }
      }
      __syncthreads();
      #pragma unroll
      for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
        #pragma unroll
        for (int ff = 0; ff < 4; ++ff) {
          #pragma unroll
          for (int yy = 0; yy < 2; ++yy) {
            compute[(((ff * 2) + yy))] = (compute[(((ff * 2) + yy))] + (pad_temp_shared[((((rc_inner * 28) + (yy * 14)) + ((int)threadIdx.x)))] * placeholder_shared[((((((int)threadIdx.z) * 32) + (ff * 8)) + rc_inner))]));
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 4; ++ax1_inner_inner_inner) {
    #pragma unroll
    for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
      T_relu[((((((((int)threadIdx.z) * 784) + (ax1_inner_inner_inner * 196)) + (((int)blockIdx.y) * 28)) + (ax2_inner_inner_inner * 14)) + ((int)threadIdx.x)))] = max((compute[(((ax1_inner_inner_inner * 2) + ax2_inner_inner_inner))] + placeholder2[(((((int)threadIdx.z) * 4) + ax1_inner_inner_inner))]), 0.000000e+00f);
    }
  }
__cuda_kelvin_exit: if (threadIdx.x == 0 & __cuda_kelvin_flag != nullptr) atomicAdd((unsigned*)__cuda_kelvin_flag, 1);
}

extern "C" __global__ void fused_nn_max_pool2d_kernel0(float* __restrict__ placeholder, float* __restrict__ tensor, volatile unsigned* __cuda_kelvin_flag) {
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int rv = 0; rv < 3; ++rv) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((int)threadIdx.x) < 256) {
        tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((int)threadIdx.x) >> 4) * 196) + (((((int)threadIdx.x) & 15) >> 2) * 42)) + (rv * 14)) + ((((int)threadIdx.x) & 3) * 3)) + rv1))]);
      }
    }
  }
  if (((int)threadIdx.x) < 256) {
    tensor[(((int)threadIdx.x))] = tensor_local[(0)];
  }
__cuda_kelvin_exit: if (threadIdx.x == 0 & __cuda_kelvin_flag != nullptr) atomicAdd((unsigned*)__cuda_kelvin_flag, 1);
}

extern "C" __global__ void fused_nn_max_pool2d_1_kernel0(float* __restrict__ placeholder, float* __restrict__ tensor, volatile unsigned* __cuda_kelvin_flag) {
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int rv = 0; rv < 2; ++rv) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 1568) {
        tensor_local[(0)] = max(tensor_local[(0)], placeholder[((((((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 14) * 56) + (rv * 28)) + ((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 14) * 2)) + rv1))]);
      }
    }
  }
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 1568) {
    tensor[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = tensor_local[(0)];
  }
__cuda_kelvin_exit: if (threadIdx.x == 0 & __cuda_kelvin_flag != nullptr) atomicAdd((unsigned*)__cuda_kelvin_flag, 1);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, volatile unsigned* __cuda_kelvin_flag) {
  float compute[4];
  __shared__ float pad_temp_shared[28];
  __shared__ float placeholder_shared[8];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((2 <= ((int)blockIdx.y)) && (2 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 58))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[(((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((2 <= ((int)blockIdx.y)) && (1 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 1))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((2 <= ((int)blockIdx.y)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 56))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 2))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((2 <= ((int)blockIdx.y)) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 27)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 55))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 3))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((2 <= ((int)blockIdx.y)) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 26)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 54))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 4))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((1 <= ((int)blockIdx.y)) && (2 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 30))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 5))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((1 <= ((int)blockIdx.y)) && (1 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 29))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 6))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((1 <= ((int)blockIdx.y)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 28))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 7))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((1 <= ((int)blockIdx.y)) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 27)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 27))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 8))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((1 <= ((int)blockIdx.y)) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 26)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 26))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 9))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((2 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 2))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 10))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((1 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) - 1))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 11))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = placeholder[((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)))];
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 12))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 27) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 1))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 13))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 26) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 2))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 14))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 27) && (2 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 26))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 15))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 27) && (1 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 27))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 16))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((((int)blockIdx.y) < 27) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 28))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 17))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 27) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 27)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 29))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 18))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 27) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 26)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 30))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 19))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 26) && (2 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 54))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 20))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 26) && (1 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 55))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 21))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = ((((int)blockIdx.y) < 26) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 56))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 22))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 26) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 27)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 23))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  __syncthreads();
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 28) {
    if (((int)threadIdx.x) < 7) {
      pad_temp_shared[(((((int)threadIdx.z) * 7) + ((int)threadIdx.x)))] = (((((int)blockIdx.y) < 26) && (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 26)) ? placeholder[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) + 58))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) < 8) {
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.z) * 2) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 50) + (((int)threadIdx.x) * 25)) + 24))];
    }
  }
  __syncthreads();
  compute[(0)] = (compute[(0)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(2)] = (compute[(2)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[((((int)threadIdx.z) * 2))]));
  compute[(1)] = (compute[(1)] + (pad_temp_shared[(((int)threadIdx.x))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  compute[(3)] = (compute[(3)] + (pad_temp_shared[((((int)threadIdx.x) + 14))] * placeholder_shared[(((((int)threadIdx.z) * 2) + 1))]));
  T_relu[((((((int)threadIdx.z) * 1568) + (((int)blockIdx.y) * 28)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[((((int)threadIdx.z) * 2))]), 0.000000e+00f);
  T_relu[(((((((int)threadIdx.z) * 1568) + (((int)blockIdx.y) * 28)) + ((int)threadIdx.x)) + 14))] = max((compute[(2)] + placeholder2[((((int)threadIdx.z) * 2))]), 0.000000e+00f);
  T_relu[(((((((int)threadIdx.z) * 1568) + (((int)blockIdx.y) * 28)) + ((int)threadIdx.x)) + 784))] = max((compute[(1)] + placeholder2[(((((int)threadIdx.z) * 2) + 1))]), 0.000000e+00f);
  T_relu[(((((((int)threadIdx.z) * 1568) + (((int)blockIdx.y) * 28)) + ((int)threadIdx.x)) + 798))] = max((compute[(3)] + placeholder2[(((((int)threadIdx.z) * 2) + 1))]), 0.000000e+00f);
__cuda_kelvin_exit: if (threadIdx.x == 0 & __cuda_kelvin_flag != nullptr) atomicAdd((unsigned*)__cuda_kelvin_flag, 1);
}

extern "C" __global__ void fused_reshape_1_kernel0(float* __restrict__ T_reshape, float* __restrict__ placeholder, volatile unsigned* __cuda_kelvin_flag) {
  if (((int)threadIdx.x) < 256) {
    T_reshape[(((int)threadIdx.x))] = placeholder[(((int)threadIdx.x))];
  }
__cuda_kelvin_exit: if (threadIdx.x == 0 & __cuda_kelvin_flag != nullptr) atomicAdd((unsigned*)__cuda_kelvin_flag, 1);
}
